#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#include "automaton.h"


// The dimension block_size of is chosen equal to 16, so that the number of
//  threads per block is a multiple of the warp size and remains below the
//  maximum number of threads per block.
#define BLOCK_SIZE 16

#define CUDA_SAFE_CALL(call) {                                               \
	hipError_t err = call;                                                    \
	if( hipSuccess != err) {                                                \
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
				__FILE__, __LINE__, hipGetErrorString( err) );              \
		exit(EXIT_FAILURE);                                                  \
	} }

struct Rule_dev
{
#define MAX_RULE_LENGTH 8
	char oldstate[MAX_RULE_LENGTH];
	char newstate[MAX_RULE_LENGTH];
	float probability;
};

__global__ void srandKernel(hiprandState* state, const size_t width,
	const int seed)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = y * width + x;
	/* Each thread gets same seed, a different sequence
	number, no offset */
	hiprand_init(seed, idx, 0, &state[idx]);
}

// This kernels runs for each row, 'height' times
__global__ void partitionHorizontalKernel(hiprandState* state,
	const size_t width, const size_t height, float omega, int* partition)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y >= height)
		return;

	for(int x = 0; x < width;)
	{
		int idx = y * width + x;
		int len = 1;
		while(float(hiprand(&state[idx])) / UINT_MAX < omega &&
			(len + x < width))
			len++;
		partition[idx] = len;
		for(int i = 1; i < len; i++)
			partition[idx+i] = 0;
		x += len;
	}
}

// This kernels runs for each column, 'width' times
__global__ void partitionVerticalKernel(hiprandState* state,
	const size_t width, const size_t height, float omega, int* partition)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= width)
		return;

	for(int y = 0; y < height;)
	{
		int idx = y * width + x;
		int len = 1;
		while(float(hiprand(&state[idx])) / UINT_MAX < omega &&
			(len + y < height))
			len++;
		partition[idx] = len;
		for(int i = 1; i < len; i++)
			partition[(y+i)*width+x] = 0;
		y += len;
	}
}

__global__ void tickHorizontalKernel(hiprandState* state, char* lattice,
	const size_t width, const size_t height,
	const Rule_dev* rules, const size_t nRules, const int* partition)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x >= width || y >= height)
		return;
	int idx = y * width + x;
	int len = partition[idx];
	if(len == 0)
		return;

	float rnd = float(hiprand(&state[idx])) / UINT_MAX;
	for(int i = 0; i < nRules; i++)
	{
		bool will_aply = true;
		int l;
		for(l = 0; l < MAX_RULE_LENGTH; l++)
		{
			if(rules[i].oldstate[l] == 0) // CUDA cannot into strlen(3)
				break;
			if(rules[i].oldstate[l] != lattice[idx+l])
			{
				will_aply = false;
				break;
			}
		}
		if(l != len)
			continue;
		if(!will_aply)
			continue;
		if(rnd < rules[i].probability)
		{
			for(int k = 0; k < l; k++)
				lattice[idx+k] = rules[i].newstate[k];
			break;
		}
	}
}

__global__ void tickVerticalKernel(hiprandState* state, char* lattice,
	const size_t width, const size_t height,
	const Rule_dev* rules, const size_t nRules, const int* partition)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x >= width || y >= height)
		return;
	int idx = y * width + x;
	int len = partition[idx];
	if(len == 0)
		return;

	float rnd = float(hiprand(&state[idx])) / UINT_MAX;
	for(int i = 0; i < nRules; i++)
	{
		bool will_aply = true;
		int l;
		for(l = 0; l < MAX_RULE_LENGTH; l++)
		{
			if(rules[i].oldstate[l] == 0) // CUDA cannot into strlen(3)
				break;
			if(rules[i].oldstate[l] != lattice[(y+l)*width + x])
			{
				will_aply = false;
				break;
			}
		}
		if(l != len)
			continue;
		if(!will_aply)
			continue;
		if(rnd < rules[i].probability)
		{
			for(int k = 0; k < l; k++)
				lattice[(y+k)*width+x] = rules[i].newstate[k];
			break;
		}
	}
}

static hiprandState* srand_dev = NULL;
static Rule_dev* rules = NULL;
static Rule_dev* rules_dev = NULL;
static size_t nrules = 0;
static char* lattice_dev = NULL;
static int* partition_dev = NULL;
static size_t size = 0;

extern "C"
__host__ void tick_cuda(struct Automaton* automaton, size_t steps)
{
	size_t k;
	Rule* r;

	int i;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE),
		dimGrid(automaton->width / dimBlock.x, automaton->height / dimBlock.y),
		dimHorizontalBlock(1, BLOCK_SIZE),
		dimHorizontalGrid(1, automaton->height / dimHorizontalBlock.y),
		dimVerticalBlock(BLOCK_SIZE, 1),
		dimVerticalGrid(automaton->width / dimVerticalBlock.x, 1);
	if(automaton->width % BLOCK_SIZE != 0)
	{
		dimGrid.x++;
		dimVerticalGrid.x++;
	}
	if(automaton->height % BLOCK_SIZE != 0)
	{
		dimGrid.y++;
		dimHorizontalGrid.y++;
	}

	if(!automaton->rules)
		return;

	automaton->ticks++;

	/* Выделить память, скопировать правила и проинициализировать ГСЧ,
	 если нужно */
	if(size < automaton->size)
	{
		if(srand_dev)
			hipFree(srand_dev);
		if(partition_dev)
			hipFree(partition_dev);
		if(lattice_dev)
			hipFree(lattice_dev);

		size = automaton->size;
		CUDA_SAFE_CALL(hipMalloc((void**)&srand_dev,
			size * sizeof(hiprandState)));
		CUDA_SAFE_CALL(hipMalloc((void**)&partition_dev,
			size * sizeof(int)));
		CUDA_SAFE_CALL(hipMalloc((void**)&lattice_dev,
			size));

		srandKernel<<<dimGrid, dimBlock>>>(srand_dev, automaton->width,
			(int)time(NULL));

		if(nrules != 0)
		{
			hipFree(rules_dev);
			hipFree(rules);
		}
		nrules = 0;
		for(r = automaton->rules; r != NULL; r = r->next)
			nrules++;
		CUDA_SAFE_CALL(hipHostMalloc((void**)&rules,
			nrules * sizeof(Rule_dev)));
		for(i = 0, r = automaton->rules; r != NULL; r = r->next, i++)
		{
			strncpy(rules[i].oldstate, r->oldstate, MAX_RULE_LENGTH);
			strncpy(rules[i].newstate, r->newstate, MAX_RULE_LENGTH);
			rules[i].probability = (float)r->probability;
		}
		CUDA_SAFE_CALL(hipMalloc((void**)&rules_dev,
			nrules * sizeof(Rule_dev)));
		CUDA_SAFE_CALL(hipMemcpy(rules_dev, rules, nrules * sizeof(Rule_dev),
			hipMemcpyHostToDevice));
	}

	hipMemcpy(lattice_dev, automaton->lattice, automaton->size,
		hipMemcpyHostToDevice);

	for(k = 0; k < steps; k++)
	{
		if(rand() > RAND_MAX/2)
		{
			partitionHorizontalKernel<<<dimHorizontalGrid, dimHorizontalBlock>>>(
				srand_dev,
				automaton->width, automaton->height,
				automaton->omega, partition_dev);
			tickHorizontalKernel<<<dimGrid, dimBlock>>>(srand_dev, lattice_dev,
				automaton->width, automaton->height,
				rules_dev, nrules, partition_dev);
		}
		else
		{
			partitionVerticalKernel<<<dimVerticalGrid, dimVerticalBlock>>>(
				srand_dev,
				automaton->width, automaton->height,
				automaton->omega, partition_dev);
			tickVerticalKernel<<<dimGrid, dimBlock>>>(srand_dev, lattice_dev,
				automaton->width, automaton->height,
				rules_dev, nrules, partition_dev);
		}
	}

	hipMemcpy(automaton->lattice, lattice_dev, automaton->size,
		hipMemcpyDeviceToHost);
}

